#include <iostream>
#include <hip/hip_runtime.h>

int check_cuda_is_avialable(void)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) 
    {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }
    else
        return 0;
}
